#include "run_cuda.cuh"
#include <fstream>

int main()
{
    std::vector<float> A(N);
    std::vector<float> B(M * N);
    std::vector<float> C0(M);
    std::vector<float> C1(M);
    std::vector<float> C2(M);
    fill_values(A, B, C0, C1, C2);
    float cuda_run_time = do_Cuda(A, B, C2);

    auto t0 = std::chrono::high_resolution_clock::now();
    vector_mx_naive(C0, A, B);
    auto t1 = std::chrono::high_resolution_clock::now();
    vector_mx_alg(C1, A, B);
    auto t2 = std::chrono::high_resolution_clock::now();
    checker(C0, C1, "C0 vs C1", "CPU Naive", "CPU improved");
    checker(C1, C2, "C0 vs C2", "CPU Improved", "GPU Naive");

    std::ofstream handler;
    std::ofstream cfg;

    handler.open("data/results.txt", std::ios_base::app);
    cfg.open("data/configs.txt", std::ios_base::app);
    handler << "CPU naive    Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count() / 1000.0f << " ms\n";
    handler << "CPU Algorithm Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0f << " ms\n";
    handler << "GPU improved Computation took: " << cuda_run_time << " ms.\n";
    handler << std::endl;
    cfg << "The N size were: " << N << ";  The M size were: " << M << std::endl;

    return 0;
}
