#include "run_cuda.cuh"
#include <fstream>

using namespace std;
int main()
{
    std::vector<float> A(N);
    std::vector<float> B(M * N);
    std::vector<float> C0(M);
    std::vector<float> C1(M);
    std::vector<float> C2(M);
    std::vector<float> C3(M);
    std::vector<float> C4(M);

    fill_values(A, B, C0, C1, C2, C3,C4);

    float gpu_1 = do_Cuda(A, B, C2);
    float gpu_2 = do_Cuda_2(A, B, C3);
    float gpu_3 = do_Cuda_3(A, B, C4);

    auto t0 = std::chrono::high_resolution_clock::now();
    vector_mx_naive(C0, A, B);
    auto t1 = std::chrono::high_resolution_clock::now();
    vector_mx_alg(C1, A, B);
    auto t2 = std::chrono::high_resolution_clock::now();
    int check_1 = checker(C0, C1, "C0 vs C1", "CPU Naive", "CPU improved");
    int check_2 = checker(C1, C2, "C0 vs C2", "CPU Improved", "GPU 1");
    int check_3 = checker(C0, C3, "C0 vs C3", "CPU Naive", "GPU 2 ");
    int check_4 = checker(C0, C4, "C0 vs C3", "CPU Naive", "GPU 2 ");


    std::ofstream handler;
    std::ofstream cfg;

    handler.open("data/results.txt", std::ios_base::app);
    cfg.open("data/configs.txt", std::ios_base::app);

    cout << "CPU naive    Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count() / 1000.0f << " ms\n";
    cout << "CPU Algorithm Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0f << " ms\n";
    cout << "GPU 1 Computation took: " << gpu_1  << " ms.\n";
    cout << "GPU 2 Computation took: " << gpu_2 << " ms.\n";
    cout << "GPU 3 Computation took: " << gpu_3  << " ms.\n";
    //if (check_1 == 0 && check_2 == 0 && check_3 == 0) { handler << "Solutions are matching" << std::endl; }
    //handler << std::endl;
    //cfg << "The N size were: " << N << ";  The M size were: " << M << std::endl;

    return 0;
}