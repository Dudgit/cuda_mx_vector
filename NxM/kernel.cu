#include "run_cuda.cuh"
#include <fstream>

int main()
{
    std::vector<float> A(N);
    std::vector<float> B(M * N);
    std::vector<float> C0(M);
    std::vector<float> C1(M);
    std::vector<float> C2(M);
    std::vector<float> C3(M);
    fill_values(A, B, C0, C1, C2,C3);
    
    float cuda_run_time = do_Cuda(A, B, C2);
    float gpu_naive = do_Cuda_naive(A, B, C3);
    
    auto t0 = std::chrono::high_resolution_clock::now();
    vector_mx_naive(C0, A, B);
    auto t1 = std::chrono::high_resolution_clock::now();
    vector_mx_alg(C1, A, B);
    auto t2 = std::chrono::high_resolution_clock::now();
    int check_1 =checker(C0, C1, "C0 vs C1", "CPU Naive", "CPU improved");
    int check_2 = checker(C1, C3, "C0 vs C2", "CPU Improved", "GPU Naive");
    int check_3 = checker(C1, C2, "C0 vs C3", "CPU Improved", "GPU Improved");


    std::ofstream handler;
    std::ofstream cfg;

    handler.open("data/results.txt", std::ios_base::app);
    cfg.open("data/configs.txt", std::ios_base::app);

    handler << "CPU naive    Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count() / 1000.0f << " ms\n";
    handler << "CPU Algorithm Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0f << " ms\n";
    handler << "GPU improved Computation took: " << cuda_run_time << " ms.\n";
    if (check_1 == 0 && check_2 == 0 && check_3 == 0) { handler << "Solutions are matching" << std::endl; }
    handler << std::endl;
    cfg << "The N size were: " << N << ";  The M size were: " << M << std::endl;

    return 0;
}
