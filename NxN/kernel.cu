#include "run_cuda.cuh"

int main()
{
    std::vector<float> A(N);
    std::vector<float> B(N * N);
    std::vector<float> C0(N);
    std::vector<float> C1(N);
    std::vector<float> C2(N);
    fill_values(A, B, C0, C1, C2);
    float cuda_run_time = do_Cuda(A, B,C2);

    auto t0 = std::chrono::high_resolution_clock::now();
    vector_mx_naive(C0, A, B, N);
    auto t1 = std::chrono::high_resolution_clock::now();
    vector_mx_alg(C1, A, B, N);
    auto t2 = std::chrono::high_resolution_clock::now();
    checker(C0, C1, "C0 vs C1","CPU Naive","CPU improved");
    checker(C0, C2, "C0 vs C2", "CPU Naive", "GPU Naive");
  

    std::cout << "CPU naive    Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count() / 1000.0f << " ms\n";
    std::cout << "CPU Algorithm Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0f << " ms\n";
    std::cout << "GPU improved Computation took: " << cuda_run_time << " ms.\n";
 
    return 0;
}
